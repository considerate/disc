#include "hip/hip_runtime.h"
#include "knn.cuh"
/*
int main(int argc, char **argv) {
    int querySize = 1 << 20;
    int dataSize = 1 << 20;
    int kSize = 70;
    int size = dataSize + querySize;
    srand(0);
    size_t valueSize = size * sizeof(float3);
    float3 *values = (float3 *) malloc(valueSize);
//  float3 values[] = {
//      {1,1,1},{2,2,2},{1,3,1},{0,1,2},{5,5,5},{3,3,3},
//      {6,6,6}, {0,0,0}, {2,3,2}, {1,2,2}   
//  };
    size_t normalSize = querySize * sizeof(float3);
    float3 *querynormals = (float3 *) malloc(normalSize);
//    float3 querynormals[] = {
//        {0,1,1},{0,1,0},{0,1,0},{0,1,0}
//    };
    size_t nearestSize = querySize * kSize * sizeof(uint64_t);
    uint64_t *nearest = (uint64_t *) malloc(nearestSize);
    initValues(values, querynormals, size, querySize);
    return nearestNeighborsEllipsoid(dataSize, querySize, kSize, values, querynormals, nearest);
}
*/

/*
// DK
int main(int argc, char **argv) {
    int numData = 20;
    int numQueries = 20;
    uint32_t k = 10;

    printf("Data,Queries,K,Queries per ms\n");
   //srand(time(NULL));
    uint64_t *nearest = NULL;
   for(int i = 0; k + i * 10 <= 160; ++i) {
       int times = (i == 0) ? 1 : 1;
       for(int j = 0; j < times; ++j) {
           int querySize = 1 << numQueries;
           int dataSize = 1 << numData;
           int kSize = k + i * 10;
           int size = dataSize + querySize;
           size_t valueSize = size * sizeof(float3);
           size_t normalSize = querySize * sizeof(float3);
           float3 *values = (float3 *) malloc(valueSize);
           float3 *querynormals = (float3 *) malloc(normalSize);
           if(argc == 3) {
               int dataPoints = readCSV(argv[1], values, 0, dataSize);
               int queryPoints = readCSV(argv[2], values, dataSize, size);
               //calculateBounds(values, dataPoints, queryPoints, dataSize, size, minx, miny, minz, maxlen);
           } else {
               initValues(values, querynormals, size, querySize);
           }
	       size_t nearestSize = querySize * kSize * sizeof(uint64_t);
	       nearest = (uint64_t *) realloc(nearest, nearestSize);
           //nearestNeighbors(dataSize, querySize, kSize, values, nearest, minx, miny, minz, maxlen);
           nearestNeighborsEllipsoid(dataSize, querySize, kSize, values, querynormals, nearest);
       }
   }
}
*/

/*
// dq
int main(int argc, char **argv) {
    int numData = 21;
    int numQueries = 7;
    uint32_t k = 64;

    printf("Data,Queries,K,Queries per ms\n");
   //srand(time(NULL));
    uint64_t *nearest = NULL;
   for(int i = 0; numQueries + i <= 21; ++i) {
       int times = (i == 0) ? 1 : 1;
       for(int j = 0; j < times; ++j) {
           int querySize = 1 << (numQueries + i);
           int dataSize = 1 << numData;
           int kSize = k;
           int size = dataSize + querySize;
           size_t valueSize = size * sizeof(float3);
           size_t normalSize = querySize * sizeof(float3);
           float3 *values = (float3 *) malloc(valueSize);
           float3 *querynormals = (float3 *) malloc(normalSize);
           if(argc == 3) {
               int dataPoints = readCSV(argv[1], values, 0, dataSize);
               int queryPoints = readCSV(argv[2], values, dataSize, size);
               //calculateBounds(values, dataPoints, queryPoints, dataSize, size, minx, miny, minz, maxlen);
           } else {
               initValues(values, querynormals, size, querySize);
           }
	       size_t nearestSize = querySize * kSize * sizeof(uint64_t);
	       nearest = (uint64_t *) realloc(nearest, nearestSize);
           //nearestNeighbors(dataSize, querySize, kSize, values, nearest, minx, miny, minz, maxlen);
           nearestNeighborsEllipsoid(dataSize, querySize, kSize, values, querynormals, nearest);
       }
   }
}
*/

// dd
int main(int argc, char **argv) {
    int numData = 7;
    int numQueries = 20;
    uint32_t k = 64;

    printf("Data,Queries,K,Queries per ms\n");
   //srand(time(NULL));
    uint64_t *nearest = NULL;
   for(int i = 0; numData + i <= 21; ++i) {
       int times = (i == 0) ? 1 : 1;
       for(int j = 0; j < times; ++j) {
           int querySize = 1 << numQueries;
           int dataSize = 1 << (numData + i);
           int kSize = k;
           int size = dataSize + querySize;
           size_t valueSize = size * sizeof(float3);
           size_t normalSize = querySize * sizeof(float3);
           float3 *values = (float3 *) malloc(valueSize);
           float3 *querynormals = (float3 *) malloc(normalSize);
           if(argc == 3) {
               int dataPoints = readCSV(argv[1], values, 0, dataSize);
               int queryPoints = readCSV(argv[2], values, dataSize, size);
               //calculateBounds(values, dataPoints, queryPoints, dataSize, size, minx, miny, minz, maxlen);
           } else {
               initValues(values, querynormals, size, querySize);
           }
	       size_t nearestSize = querySize * kSize * sizeof(uint64_t);
	       nearest = (uint64_t *) realloc(nearest, nearestSize);
           //nearestNeighbors(dataSize, querySize, kSize, values, nearest, minx, miny, minz, maxlen);
           nearestNeighborsEllipsoid(dataSize, querySize, kSize, values, querynormals, nearest);
       }
   }
}
